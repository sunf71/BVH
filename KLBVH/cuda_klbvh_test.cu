#include "hip/hip_runtime.h"
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include "cuda_klbvh.h"
#include "MortonCode.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "objLoader.h"
#include <algorithm>
#include <cstdlib>
#include "klbvh.h"
#include "gputimer.cuh"
#include "timer.h"
#include <hipcub/hipcub.hpp>
using namespace nih;
using namespace cuda;
int globalCounter = 0;
//����bvhʱ������ͼԪ���
uint32* gd_indices;
//bvh
Bintree_node* gd_nbvh;

texture<float> BboxTex;
texture<uint32> bvhTex;
//����BVHʱ������ͼԪ����
texture<uint32> indexTex;

__constant__ uint32 tx[2];
__constant__ uint32 ty[2];
__constant__ uint32 tz[2];
__constant__ uint32 ttx[2];
__constant__ uint32 tty[2];
__constant__ uint32 ttz[2];

struct cullingContext
{
	__forceinline__  __host__ __device__ cullingContext()
	{
		triId = uint32(-1);
	}
	__forceinline__ __host__ __device__ cullingContext(uint32 f, uint32 t)
	{
		frustumId = f;
		triId = t;
	}
	uint32 frustumId;
	uint32 triId;
};
struct is_valid
{
	__forceinline__ __host__ __device__ bool operator()(const cullingContext& c)
	{
		return c.triId != uint32(-1);
	}
};
uint32 GridSize(uint32 count)
{
	int numSMs;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
	size_t max_blocks = 65535;
	size_t n_blocks   = nih::min( max_blocks, (count + (128*numSMs)-1) / (128*numSMs) );
	return numSMs*n_blocks;
}

struct bvhTexHelper
{
	static const uint32 nodeSize = 11;
	static const uint32 LChildOf = 6;
	static const uint32 RChildOf = 7;
	static const uint32 pidOf = 8;
	static const uint32 leafStartOf = 9;
	static const uint32 leafEndOf = 10;
	float p[6];
	NIH_DEVICE float* getBbox(uint32 id)
	{
		uint32 offset = id*nodeSize;
		uint32 t[6];

		for(int i=0; i<6; i++)
		{
			t[i] = tex1Dfetch(bvhTex,offset+i);
			p[i] = bitsToFloat(t[i]);
		}


		return p;
	}

	NIH_DEVICE uint32 getLChild(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+LChildOf);
	}

	NIH_DEVICE uint32 getRChild(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+RChildOf);
	}
	NIH_DEVICE uint32 getPid(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+pidOf);
	}
	NIH_DEVICE uint32 getleafStart(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+leafStartOf);
	}
	NIH_DEVICE uint32 getleafEnd(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+leafEndOf);
	}
	NIH_DEVICE bool isLeaf(uint32 id)
	{
		return getleafStart(id) == getleafEnd(id);
	}
};

//FORCE_INLINE NIH_DEVICE void traverseIterative(Bbox3f& qbox, uint32 qId,uint32 leafIdx,
//	cuda::DBVH* bvh,
//	uint32* list)
//{
//	Bvh_Node* stack[64];
//	uint32 top = 0;
//	stack[top++] = bvh->getRoot();
//	while(top>0)
//	{
//		Bvh_Node* node = stack[--top];
//		if (node->leafEnd <= leafIdx)
//			continue;
//		Bbox3f box = bvh->getNodeBox(node);
//		
//		bool ret = AABBOverlap(qbox, box);
//		if (ret )
//		{
//			//�ཻ
//			if (node->l_isleaf)
//			{
//				Bvh_Node* leaf = (bvh->getLLeafChild(node));
//				Bbox3f box = bvh->getLeafBox(leaf);
//				if (AABBOverlap(qbox,box))
//				{
//					list->add(Pair(qId,leaf->pid));
//				}				
//			}
//			else
//				stack[top++] = bvh->getLChild(node);
//			if(node->r_isleaf)
//			{
//				Bvh_Node* leaf = (bvh->getRLeafChild(node));
//				Bbox3f box = bvh->getLeafBox(leaf);
//				if (AABBOverlap(qbox,box))
//				{
//					list->add(Pair(qId,leaf->pid));
//				}				
//			}
//			else
//				stack[top++] =  bvh->getRChild(node);
//		}		
//	}
//}
FORCE_INLINE NIH_DEVICE void FrustumCulling(pyrfrustum_t& frustum, uint32 frustumId,
	cuda::DBVH* bvh,uint32 priSize,
	uint32* list)
{
	Bvh_Node* stack[64];
	uint32 top = 0;
	stack[top++] = bvh->getRoot();
	uint32 offset = priSize*frustumId;
	while(top>0)
	{
		Bvh_Node* node = stack[--top];
		Bbox3f box = bvh->getNodeBox(node);

		int ret = Intersect(frustum, box);
		if (ret == 2)
		{
			//�ཻ
			if (node->l_isleaf)
			{
				Bvh_Node* leaf = (bvh->getLLeafChild(node));
				Bbox3f box = bvh->getLeafBox(leaf);
				if (Intersect(frustum,box))
				{
					list[offset+leaf->pid] = 1;
				}				
			}
			else
				stack[top++] = bvh->getLChild(node);
			if(node->r_isleaf)
			{
				Bvh_Node* leaf = (bvh->getRLeafChild(node));
				Bbox3f box = bvh->getLeafBox(leaf);
				if (Intersect(frustum,box))
				{
					list[offset+leaf->pid] = 1;
				}				
			}
			else
				stack[top++] =  bvh->getRChild(node);
		}
		else if (ret == 1)
		{
			//in
			for(int k= node->leafStart; k<=node->leafEnd;k++)
				list[offset+bvh->leaves[k].pid] = 1;
		}
	}
}
FORCE_INLINE NIH_DEVICE void FrustumCulling(TriFrustum& frustum, uint32 frustumId,
	cuda::DBVH* bvh,uint32 priSize,
	cullingContext* list)	
{
	Bvh_Node* stack[64];
	uint32 top = 0;
	stack[top++] = bvh->getRoot();
	uint32 offset = priSize*frustumId;
	while(top>0)
	{
		Bvh_Node* node = stack[--top];
		Bbox3f box = bvh->getNodeBox(node);

		int ret = Intersect(frustum, box);
		if (ret == 2)
		{
			//�ཻ
			if (node->l_isleaf)
			{
				Bvh_Node* leaf = (bvh->getLLeafChild(node));
				if (leaf->pid<priSize)
				{
					Bbox3f box = bvh->getLeafBox(leaf);
					if (Intersect(frustum,box))
					{
						list[offset+leaf->pid].triId = leaf->pid;
						list[offset+leaf->pid].frustumId = frustumId;
					}				
				}
			}
			else
				stack[top++] = bvh->getLChild(node);
			if(node->r_isleaf)
			{
				Bvh_Node* leaf = (bvh->getRLeafChild(node));
				if (leaf->pid < priSize)
				{
					Bbox3f box = bvh->getLeafBox(leaf);
					if (Intersect(frustum,box))
					{
						list[offset+leaf->pid].triId = leaf->pid;
						list[offset+leaf->pid].frustumId = frustumId;
					}	
				}
			}
			else
				stack[top++] =  bvh->getRChild(node);
		}
		else if (ret == 1)
		{
			//in
			for(int k= node->leafStart; k<=node->leafEnd;k++)
			{
				if(bvh->leaves[k].pid<priSize)
				{
					list[offset+bvh->leaves[k].pid].triId = bvh->leaves[k].pid;
					list[offset+bvh->leaves[k].pid].frustumId = frustumId;
				}
			}
		}
	}
}
FORCE_INLINE NIH_DEVICE void FrustumCulling(pyrfrustum_t& frustum, uint32 frustumId,
	Bintree* bvh,uint32 priSize,
	uint32* out)
{
	uint32 offset = priSize*frustumId;
	const uint32 stack_size  = 64;
	uint32 stack[stack_size];
	uint32 top = 0;
	stack[top++] = 0;
	while(top>0)
	{
		uint32 idx = stack[--top];		
		int ret = Intersect(frustum,bvh->boxPtr[idx]);
		if (ret == 2)
		{
			//�ཻ

			if(bvh->isLeafPtr[bvh->RChildPtr[idx]])
			{
				if (Intersect(frustum,bvh->boxPtr[bvh->RChildPtr[idx]]))
				{
					out[offset+bvh->pidPtr[bvh->RChildPtr[idx]]] = 1;					
				}				
			}
			else
				stack[top++] = bvh->RChildPtr[idx];

			if (bvh->isLeafPtr[bvh->LChildPtr[idx]])
			{

				if (Intersect(frustum,bvh->boxPtr[bvh->LChildPtr[idx]]))
				{
					out[offset+bvh->pidPtr[bvh->LChildPtr[idx]]] = 1;					
				}				
			}
			else
				stack[top++] = bvh->LChildPtr[idx];
		}
		else if (ret == 1)
		{
			//in
			for(int k= bvh->leafStartPtr[idx]; k<=bvh->leafEndPtr[idx];k++)
				out[offset+offset+bvh->pidPtr[k]] = 1;

		}
	}
}
FORCE_INLINE NIH_DEVICE void FrustumCulling(pyrfrustum_t& frustum, uint32 frustumId,
	Bintree_Node* bvh,uint32 priSize,
	uint32* out)
{
	uint32 offset = priSize*frustumId;
	const uint32 stack_size  = 64;
	Bintree_Node* stack[stack_size];
	Bintree_Node** stackPtr = stack;
	*stackPtr++ = NULL;
	Bintree_Node* node = &bvh[0];
	do
	{
		// Check each child node for overlap.
		Bintree_Node* childL = &bvh[node->lChild];
		Bintree_Node* childR = &bvh[node->RChild];
		int overlapL = ( Intersect(frustum, 
			node->lBox) );
		int overlapR = ( Intersect(frustum, 
			node->rBox) );

		// Query overlaps a leaf node => report collision.
		if (overlapL && bvh[node->lChild].isLeaf())
			out[offset + bvh[node->lChild].leafStart] = 1;

		if (overlapR && bvh[node->RChild].isLeaf())
			out[offset + bvh[node->RChild].leafStart] = 1;

		// Query overlaps an internal node => traverse.
		bool traverseL = false;
		if (overlapL == 1)
		{
			for(int k= childL->leafStart; k<=childL->leafEnd; k++)
				out[offset + k] = 1;
		}
		else if( overlapL == 2 && !bvh[node->lChild].isLeaf())
		{
			traverseL = true;
		}
		bool traverseR = false;
		if (overlapR == 1)
		{
			for(int k= childR->leafStart; k<=childR->leafEnd; k++)
				out[offset + k] = 1;
		}
		else if( overlapR == 2 && !bvh[node->RChild].isLeaf())
		{
			traverseR = true;
		}

		if (!traverseL && !traverseR)
			node = *--stackPtr; // pop
		else
		{
			node = (traverseL) ? childL : childR;
			if (traverseL && traverseR)
				*stackPtr++ = childR; // push
		}
	}
	while (node != NULL);
}
FORCE_INLINE NIH_DEVICE void FrustumCulling(pyrfrustum_t& frustum, uint32 frustumId,
	Bintree_node* bvh,uint32 priSize,
	uint32* out)
{
	bvhTexHelper helper;
	uint32 offset = priSize*frustumId;
	const uint32 stack_size  = 64;
	uint32 stack[stack_size];
	uint32 top = 0;
	stack[top++] = 0;
	while(top>0)
	{
		uint32 idx = stack[--top];
		//Bintree_node * node = &bvh[idx];
		uint32 RChild = helper.getRChild(idx);
		uint32 LChild = helper.getLChild(idx);

		int ret = Intersect(frustum,helper.getBbox(idx));
		if (ret == 2)
		{
			//�ཻ

			if(helper.isLeaf(RChild))
			{
				if (Intersect(frustum,helper.getBbox(RChild)))
				{
					out[offset+helper.getPid(RChild)] = 1;					
				}				
			}
			else
				stack[top++] = RChild;

			if (helper.isLeaf(LChild))
			{

				if (Intersect(frustum,helper.getBbox(LChild)))
				{
					out[offset+helper.getPid(LChild)] = 1;					
				}				
			}
			else
				stack[top++] = LChild;
		}
		else if (ret == 1)
		{
			//in
			for(int k= helper.getleafStart(idx); k<=helper.getleafEnd(idx);k++)
				out[offset+k] = 1;
		}
	}
}


//FORCE_INLINE NIH_DEVICE void FrustumCulling(pyrfrustum_t& frustum, uint32 frustumId,
//	uint32 priSize,
//	uint32* out)
//{
//	uint32 offset = priSize*frustumId;
//	const uint32 stack_size  = 64;
//	const uint32 nodeOffset = 64;
//	uint32 stack[stack_size];
//	uint32 top = 0;
//	stack[top++] = 0;
//	while(top>0)
//	{
//		uint32 idx = stack[--top];
//		Bintree_node * node = tex1Dfetch(bvhTex,idx*64);
//		int ret = Intersect(frustum,node->box);
//		if (ret == 2)
//		{
//			//�ཻ
//			
//			if(bvh[node->RChild].isLeaf())
//			{
//				if (Intersect(frustum,bvh[node->RChild].box))
//				{
//					out[offset+bvh[node->RChild].pid] = 1;					
//				}				
//			}
//			else
//				stack[top++] = node->RChild;
//
//			if (bvh[node->lChild].isLeaf())
//			{
//				
//				if (Intersect(frustum,bvh[node->lChild].box))
//				{
//					out[offset+bvh[node->lChild].pid] = 1;					
//				}				
//			}
//			else
//				stack[top++] = node->lChild;
//		}
//		else if (ret == 1)
//		{
//			//in
//			for(int k= node->leafStart; k<=node->leafEnd;k++)
//				out[offset+k] = 1;
//		}
//	}
//}
//FORCE_INLINE NIH_DEVICE void FrustumCullingT(pyrfrustum_t& frustum, uint32 frustumId,
//	Bintree_node* bvh,uint32 priSize,
//	uint32* out)
//{
//	Bintree_node* stack[64];
//    Bintree_node** stackPtr = stack;
//    *stackPtr++ = NULL; // push
//	uint32 offset = priSize*frustumId;
//    // Traverse nodes starting from the root.
//    Bintree_node* node = &bvh[0];
//    do
//    {
//        // Check each child node for overlap.
//		Bintree_node* childL = &bvh[node->lChild];
//        Bintree_node* childR = &bvh[node->RChild];
//        int overlapL = ( Intersect(frustum, 
//			childL->box) );
//        int overlapR = (Intersect(frustum, 
//			childR->box) );
//
//        // Query overlaps a leaf node => report collision.
//		if (overlapL>0 && childL->isLeaf())
//			out[offset+ childL->pid] = 1;
//
//		if (overlapR>0 && childR->isLeaf())
//            out[offset+ childR->pid] = 1;
//
//		/*if (overlapL == 1 && !childL->isLeaf)
//		{
//			for(int k=childL->leafStart; k<=childL->leafEnd;k++)
//				out[offset+k] = 1;
//		}
//
//		if (overlapR == 1 && !childR->isLeaf )
//		{
//			for(int k=childR->leafStart; k<=childR->leafEnd;k++)
//				out[offset+k] = 1;
//		}*/
//        // Query overlaps an internal node => traverse.
//        bool traverseL = (overlapL == 2 && !childL->isLeaf());
//        bool traverseR = (overlapR ==2 && !childR->isLeaf());
//
//        if (!traverseL && !traverseR)
//            node = *--stackPtr; // pop
//        else
//        {
//            node = (traverseL) ? childL : childR;
//            if (traverseL && traverseR)
//                *stackPtr++ = childR; // push
//        }
//    }
//    while (node != NULL);
//}
FORCE_INLINE NIH_HOST_DEVICE int IntersectSmart( pyrfrustum_t& f, float*ptr)
{
	bool intersec = false;
	float*p = ptr;
	for(int i=0; i<6; i++)
	{
		//plane_t plane= f.planes[i];
		uint32 sa = f.planes[i].a > 0; 
		uint32 sb = f.planes[i].b >0; 
		uint32 sc = f.planes[i].c > 0;
		if (p[tx[sa]]*f.planes[i].a + p[ty[sb]]*f.planes[i].b + p[tz[sc]]*f.planes[i].c+f.planes[i].d <=0)
			return 0;
		if (p[ttx[sa]]*f.planes[i].a + p[tty[sb]]*f.planes[i].b + p[ttz[sc]]*f.planes[i].c+f.planes[i].d <=0)
			intersec = true;

	}
	return intersec ? 2 : 1; 
}
FORCE_INLINE NIH_HOST_DEVICE int IntersectSmart( TriFrustum& f, Bbox3f& box )
{

	bool intersec = false;
	float*p = &box.m_min[0];
	for(int i=0; i<5; i++)
	{
		//plane_t plane= f.planes[i];
		uint32 sa = signbit(f.planes[i].a); 
		uint32 sb = signbit(f.planes[i].b);
		uint32 sc = signbit(f.planes[i].c);
		
		if (p[tx[sa]]*f.planes[i].a + p[ty[sb]]*f.planes[i].b + p[tz[sc]]*f.planes[i].c+f.planes[i].d > 0)
			return 0;
		if (p[ttx[sa]]*f.planes[i].a + p[tty[sb]]*f.planes[i].b + p[ttz[sc]]*f.planes[i].c+f.planes[i].d > 0)
			intersec = true;

	}
	return intersec ? 2 : 1; 
}
FORCE_INLINE NIH_HOST_DEVICE int IntersectSmart( TriFrustum& f, float*p )
{

	bool intersec = false;

	for(int i=0; i<5; i++)
	{
		//plane_t plane= f.planes[i];
		uint32 sa = signbit(f.planes[i].a); 
		uint32 sb = signbit(f.planes[i].b);
		uint32 sc = signbit(f.planes[i].c);
		if (p[tx[sa]]*f.planes[i].a + p[ty[sb]]*f.planes[i].b + p[tz[sc]]*f.planes[i].c+f.planes[i].d > 0)
			return 0;
		if (p[ttx[sa]]*f.planes[i].a + p[tty[sb]]*f.planes[i].b + p[ttz[sc]]*f.planes[i].c+f.planes[i].d > 0)
			intersec = true;

	}
	return intersec ? 2 : 1; 
}
__global__ void BruteforceFrustumCullingKernel(TriFrustum* frustum, uint32 frustumNum, Bbox3f* boxes, uint32 boxNum,cullingContext* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < boxNum; 
		i += step) 
	{
		for(int j=0; j<frustumNum; j++)
		{
			if (IntersectSmart(frustum[j],boxes[i]))
			{
				uint32 offset = boxNum*j+i;
				list[offset].frustumId = frustum[j].id;
				list[offset].triId = i;
			}
		}
	}
}

__global__ void BruteforceFrustumCullingKernel(pyrfrustum_t* frustum, Bbox3f* boxes, uint32 priSize,uint32* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < priSize; 
		i += step) 
	{
		int offset = i*6;
		float p[6];
		p[0] = tex1Dfetch(BboxTex,offset);
		p[1] = tex1Dfetch(BboxTex,offset+1);
		p[2] = tex1Dfetch(BboxTex,offset+2);
		p[3] = tex1Dfetch(BboxTex,offset+3);
		p[4] = tex1Dfetch(BboxTex,offset+4);
		p[5] = tex1Dfetch(BboxTex,offset+5);


		if (IntersectSmart(*frustum,p) >0 )
			list[i] = 1;
	}
}
__global__ void FrustumCullingKernel(pyrfrustum_t* frustumP, int frustum_num, Bintree_Node* bvh, uint32 priSize,uint32* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < frustum_num; 
		i += step) 
	{
		pyrfrustum_t frustum = frustumP[i];
		FrustumCulling(frustum,i,bvh,priSize,list);
		//FrustumCullingT(frustum,i,bvh,priSize,list);
	}
}
__global__ void FrustumCullingKernel(pyrfrustum_t* frustumP, int frustum_num, Bintree_node* bvh, uint32 priSize,uint32* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < frustum_num; 
		i += step) 
	{
		pyrfrustum_t frustum = frustumP[i];
		FrustumCulling(frustum,i,bvh,priSize,list);
		//FrustumCullingT(frustum,i,bvh,priSize,list);
	}
}
__global__ void FrustumCullingKernel(pyrfrustum_t* frustumP, int frustum_num, Bintree* bvh, uint32 priSize,uint32* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < frustum_num; 
		i += step) 
	{
		pyrfrustum_t frustum = frustumP[i];
		FrustumCulling(frustum,i,bvh,priSize,list);
		//FrustumCullingT(frustum,i,bvh,priSize,list);
	}
}
__global__ void FrustumCullingKernel(TriFrustum* frustumP, int frustum_num, cuda::DBVH* bvh, uint32 priSize,cullingContext* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < frustum_num; 
		i += step) 
	{
		TriFrustum frustum = frustumP[i];
		FrustumCulling(frustum,i,bvh,priSize,list);
	}
}
__global__ void FrustumCullingKernel(pyrfrustum_t* frustumP, int frustum_num, cuda::DBVH* bvh, uint32 priSize,uint32* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < frustum_num; 
		i += step) 
	{
		pyrfrustum_t frustum = frustumP[i];
		FrustumCulling(frustum,i,bvh,priSize,list);
	}
}

bool NIH_HOST_DEVICE GenerateVirFrustum(uint32 id, const Vector3f& eye,const Vector3f& p1,const Vector3f& p2, const Vector3f& p3, float farD, TriFrustum& frustum)
{
	//��5��ƽ�淽��
	//��׶ƽ�淨��ָ����׶��
	plane_t pTri(p1,p2,p3);	

	float d  = pTri.distance(eye);
	//�ӵ㲻��λ��������ƽ�淨����һ��
	if (d<= 0)
		return false;

	//�����ӵ�
	Vector3f fNormal(pTri.a,pTri.b,pTri.c);
	float dir = dot(eye-p1,fNormal);
	Vector3f vEye = eye-fNormal*2.f*dir;

	frustum.id = id;
	frustum.planes[0] = plane_t(vEye,p2,p1);
	frustum.planes[1] = plane_t(vEye,p3,p2);
	frustum.planes[2] = plane_t(vEye,p1,p3);
	frustum.planes[3] =  plane_t(p1,p3,p2);
	frustum.planes[4] = pTri;
	Vector3f c = (p1+p2+p3)*1.f/3.f;
	float cosT = d/euclidean_distance(vEye,c);
	frustum.planes[4].d -= farD*cosT;		
	Vector3f p4,p5,p6;
	frustum.planes[4].intersect(vEye,p1,p4);
	frustum.planes[4].intersect(vEye,p2,p5);
	frustum.planes[4].intersect(vEye,p3,p6);
	frustum.center = (p1+p2+p3+p4+p5+p6)*1.f/6.f;
	frustum.min = nih::min(p1,p2);
	frustum.min = nih::min(frustum.min,p3);
	frustum.min = nih::min(frustum.min,p4);
	frustum.min = nih::min(frustum.min,p5);
	frustum.min = nih::min(frustum.min,p6);

	frustum.max = nih::max(p1,p2);
	frustum.max = nih::max(frustum.max,p3);
	frustum.max = nih::max(frustum.max,p4);
	frustum.max = nih::max(frustum.max,p5);
	frustum.max = nih::max(frustum.max,p6);
	return true;
}

//����׶����kernel
__global__ void GenerateVirFrustumKernel(Vector3f* eye,Vector3f* p123, TriFrustum* frustums,float farD, int count)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < count; 
		i += step) 
	{
		GenerateVirFrustum(i,*eye,p123[i*3],p123[i*3+1],p123[i*3+2],farD,frustums[i]);

	}
}

//��Χ�вü��������ڰ�Χ���ڵ��������б�
//@Box ��Χ��
//@TrianglePoints �����Σ�ÿ��������3���㣩
//@size ����������
//@list �ڰ�Χ���ڵ�������id
__global__ void BboxCullingKernel(Bbox3f* Box, Vector3f* TrianglePoints, uint32 size, char* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < size; 
		i += step) 
	{
		uint32 offset = 3*i;
		uint32 in = 0;
		for(int j=0; j<3; j++)
		{
			Vector3f p = TrianglePoints[offset+j];
			if (contains(*Box,p))
				in++;
		}

		if (in > 0)
		{
			list[offset] = 1;
			list[offset+1] = 1;
			list[offset+2] = 1;

		}

	}
}

__global__ void PrepareBvhPointKernel(Vector3f* TrianglePoints, Vector3f* centers,Bbox3f* boxes, uint32 size)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < size; 
		i += step) 
	{
		uint32 offset = i*3;
		centers[i] = (TrianglePoints[offset] +TrianglePoints[offset+1] +TrianglePoints[offset+2])/3.f;
		boxes[i] = Bbox3f(TrianglePoints[offset]);
		boxes[i].insert(TrianglePoints[offset+1]);
		boxes[i].insert(TrianglePoints[offset+2]);

	}
}
__global__ void PrepareBvhPointKernel(TriFrustum* frustums, Vector3f* centers,Bbox3f* boxes, uint32 size)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < size; 
		i += step) 
	{

		centers[i] = frustums[i].center;
		boxes[i] = Bbox3f(frustums[i].min,frustums[i].max);


	}
}
FORCE_INLINE NIH_DEVICE void FrustumCulling(TriFrustum& frustum, uint32 frustumId, uint32 idx,
	uint32 priSize,
	cullingContext* out)
{
	bvhTexHelper helper;
	uint32 offset = priSize*idx;
	const uint32 stack_size  = 64;
	uint32 stack[stack_size];
	uint32 top = 0;
	stack[top++] = 0;
	while(top>0)
	{
		uint32 idx = stack[--top];
		//Bintree_node * node = &bvh[idx];
		uint32 RChild = helper.getRChild(idx);
		uint32 LChild = helper.getLChild(idx);

		int ret = IntersectSmart(frustum,helper.getBbox(idx));
		if (ret == 2)
		{
			//�ཻ
			if(helper.isLeaf(RChild))
			{
				if (IntersectSmart(frustum,helper.getBbox(RChild)))
				{
					uint32 pid = helper.getPid(RChild);
					pid = tex1Dfetch(indexTex,pid);
					out[offset+pid].frustumId = frustumId;
					out[offset+pid].triId = pid;					
				}				
			}
			else
				stack[top++] = RChild;

			if (helper.isLeaf(LChild))
			{

				if (IntersectSmart(frustum,helper.getBbox(LChild)))
				{
					uint32 pid = helper.getPid(LChild);
					pid = tex1Dfetch(indexTex,pid);
					out[offset+pid].frustumId = frustumId;					
					out[offset+pid].triId = pid;
				}				
			}
			else
				stack[top++] = LChild;
		}
		else if (ret == 1)
		{
			//in
			for(int k= helper.getleafStart(idx); k<=helper.getleafEnd(idx);k++)
			{						
				uint32	pid = tex1Dfetch(indexTex,k);
				out[offset+pid].frustumId = frustumId;
				out[offset+pid].triId = pid;
			}
		}
	}
}



__global__ void FrustumCullingKernel(TriFrustum* frustumP, int frustum_num, uint32 priSize,cullingContext* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < frustum_num; 
		i += step) 
	{
		TriFrustum frustum = frustumP[i];
		FrustumCulling(frustum,frustumP[i].id,i,priSize,list);
		//FrustumCullingT(frustum,i,bvh,priSize,list);
	}
}
__global__ void FrustumCullingKernel(TriFrustum* frustums,cuda::DBVH* d_bvh,uint32 frustumNum, uint32 leavesNum,cullingContext* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < leavesNum; 
		i += step) 
	{
		if (d_bvh->leaves[i].pid >= leavesNum)
		{
			TriFrustum frustum = frustums[d_bvh->leaves[i].pid - leavesNum];
			FrustumCulling(frustum,d_bvh->leaves[i].pid - leavesNum,d_bvh,leavesNum,list);
		}
	}
}
//__global__ void CollisonDetectionKernel(cuda::DBVH* bvh, cuda::SimpleList<Pair>* list, int size)
//{
//	int idx = threadIdx.x + blockDim.x * blockIdx.x;
//    if (idx < size)
//    {
//        Bbox3f qbox = bvh->leafBoxes[idx];
//		uint32 qId = bvh->leaves[idx].pid;
//        traverseIterative(qbox,qId, idx, bvh,list); 
//                      
//    }
//}



void CpuKlbvhTest(const thrust::host_vector<Vector3f>& h_points,
	const thrust::host_vector<Bbox3f>& h_boxes, const Bbox3f& gbox,
	pyrfrustum_t& frustum, BVH& bvh);
void GpuKlbvhTest(const thrust::host_vector<Vector3f>& h_points,
	const thrust::host_vector<Bbox3f>& h_boxes, 
	const Bbox3f& gBox,
	pyrfrustum_t& frustum, 
	thrust::host_vector<Bvh_Node>& h_nodes,
	thrust::host_vector<Bvh_Node>& h_leaves,
	thrust::host_vector<Bbox3f>& h_nodeBoxes,
	thrust::host_vector<Bbox3f>& h_leafBoxes
	)
{

	size_t size = h_points.size();
	thrust::device_vector<Vector3f> d_points = h_points;
	thrust::device_vector<Bbox3f> d_boxes = h_boxes;

	//{
	//random data
	/*size = 12;
	thrust::host_vector<Vector3f> points(size);
	thrust::host_vector<Bbox3f> boxes(size);

	for(int i =0; i< points.size(); i++)
	{
	points[i] = Vector3f(1.0*(rand()%100)/100,1.0*(rand()%100)/100,1.0*(rand()%100)/100);
	printVector3f(points[i]);
	boxes[i].insert(points[i]);
	gBox.insert(boxes[i]);
	}	
	d_points = points;
	d_boxes = boxes;*/
	//}
	//load from obj file
	/*size = loadObj(objFileName,d_points,d_boxes,gBox);*/
	thrust::device_vector<Bvh_Node> nodes(size-1);
	thrust::device_vector<Bvh_Node> leaves(size);
	hipcub::CachingDeviceAllocator allocator(true);
	cuda::KBvh_Builder builder(nodes,leaves,allocator);

	/*hipcub::DoubleBuffer<uint32> d_codes;
	size_t n_points = points.size();
	allocator.DeviceAllocate((void**)&d_codes.d_buffers[0], sizeof(uint32) * n_points);
	thrust::device_ptr<uint32> d_ptr(d_codes.d_buffers[0]);
	thrust::transform(
	d_points.begin(),
	d_points.begin() + n_points,
	d_ptr,
	morton_functor<uint32>( gBox ) );
	uint32 * h_codes = (uint32*)malloc(sizeof(uint32)*n_points);
	hipMemcpy(h_codes,d_codes.d_buffers[0],sizeof(uint32)*n_points,hipMemcpyDeviceToHost);
	for(int i=0; i<n_points; i++)
	std::cout<<h_codes[i]<<std::endl;*/
	cuda::DBVH h_bvh;
	/*hipMalloc((void**)&d_bvh,sizeof(cuda::DBVH));*/
	GpuTimer timer;
	timer.Start();
	builder.build(gBox,d_points.begin(),d_points.end(),d_boxes.begin(),d_boxes.end(),&h_bvh);
	timer.Stop();
	std::cout<<"build time "<<timer.ElapsedMillis()<<" ms"<<std::endl;

	hipBindTexture( NULL, BboxTex,
		builder.getLeafBoxes(),
		sizeof(Bbox3f)*size );

	h_nodes = nodes;
	h_leaves = leaves;	


	//for(int i = 0; i<h_nodes.size(); i++)
	//{ 
	//	std::cout<<" parent idx is "<<h_nodes[i].parentIdx<<" ,";
	//	
	//	if(h_nodes[i].l_isleaf)
	//	{
	//		std::cout<<i<<" left child "<<" is leaf "<<h_nodes[i].getChild(0);
	//	}
	//	else
	//	{
	//		std::cout<<i<<" left child "<<" is internal "<<h_nodes[i].getChild(0);				

	//	}
	//	if(h_nodes[i].r_isleaf)
	//	{
	//		std::cout<<" right child "<<" is leaf "<<h_nodes[i].getChild(1)<<std::endl;
	//	}
	//	else
	//	{
	//		std::cout<<" right child "<<" is internal "<<h_nodes[i].getChild(1)<<std::endl;
	//	}
	//}
	//for(int i=0; i<h_leaves.size(); i++)
	//{
	//	std::cout<<i<<" parent is "<<h_leaves[i].parentIdx<<std::endl;
	//	std::cout<<" pid is "<<h_leaves[i].pid<<std::endl;
	//}

	h_nodeBoxes.resize(size-1);
	Bbox3f* p_nodeBoxes = thrust::raw_pointer_cast(&h_nodeBoxes.front());
	hipMemcpy(p_nodeBoxes,builder.getNodeBoxes(),sizeof(Bbox3f)*(size-1),hipMemcpyDeviceToHost);

	h_leafBoxes.resize(size);
	Bbox3f* p_leafBoxes = thrust::raw_pointer_cast(&h_leafBoxes.front());
	hipMemcpy(p_leafBoxes,builder.getLeafBoxes(),sizeof(Bbox3f)*(size),hipMemcpyDeviceToHost);




	int frustumCount = 1;
	thrust::host_vector<pyrfrustum_t> h_frustums(frustumCount);
	for(int i=0; i< frustumCount; i++)
		h_frustums[i] = frustum;

	const uint32 BLOCK_SIZE = 128;
	int numSMs;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
	size_t max_blocks = 65535;
	size_t n_blocks   = nih::min( max_blocks, (frustumCount + (BLOCK_SIZE*numSMs)-1) / (BLOCK_SIZE*numSMs) );
	thrust::device_vector<pyrfrustum_t> d_frustums = h_frustums;

	Bvh_Node* d_nodes = thrust::raw_pointer_cast(&(nodes.front()));
	Bvh_Node* d_leaves = thrust::raw_pointer_cast(&(leaves.front()));
	cuda::DBVH* d_bvh = NULL;
	hipMalloc((void**)&d_bvh,sizeof(BVH));
	hipMemcpy(d_bvh,&h_bvh,sizeof(BVH),hipMemcpyHostToDevice);
	uint32* d_list;
	thrust::device_vector<uint32> d_vector(size*frustumCount,0);
	d_list = thrust::raw_pointer_cast(&d_vector.front());
	timer.Start();
	FrustumCullingKernel<<<n_blocks*numSMs,BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_frustums.front()),frustumCount, d_bvh, size,d_list);
	timer.Stop();
	std::cout<<"culling time "<<timer.ElapsedMillis()<<" ms"<<std::endl;
	thrust::host_vector<uint32> h_list = d_vector;
	uint32 k = 0;
	for(int i=0; i<size*frustumCount; i++)
		if(h_list[i] == 1)
			k++;
	std::cout<<"total "<<h_list.size()<<" in "<<k<<std::endl;



	Bintree_node* nbvh;
	uint32 nbvh_size = h_nodes.size()+h_leaves.size();
	nbvh = new Bintree_node[nbvh_size];
	Timer ctimer;
	ctimer.start();
	cuda::DFSBintree(&h_nodes,&h_leaves,&h_nodeBoxes,&h_leafBoxes,nbvh);
	//cuda::BFSBintree(&h_nodes,&h_leaves,&h_nodeBoxes,&h_leafBoxes,nbvh);
	ctimer.stop();
	std::cout<<"build DFSBintree time "<<ctimer.seconds()*1000<<"ms"<<std::endl;
	Bintree_node* d_nbvh;
	hipMalloc((void**)&d_nbvh,sizeof(Bintree_node)*nbvh_size);
	hipMemcpy(d_nbvh,nbvh,sizeof(Bintree_node)*nbvh_size,hipMemcpyHostToDevice);
	hipBindTexture( NULL, bvhTex,
		d_nbvh, sizeof(Bintree_node)*nbvh_size );

	thrust::device_vector<uint32> d_vector1(size*frustumCount,0);
	d_list = thrust::raw_pointer_cast(&d_vector1.front());
	timer.Start();
	FrustumCullingKernel<<<n_blocks*numSMs,BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_frustums.front()),frustumCount, d_nbvh, size,d_list);
	timer.Stop();
	std::cout<<"dfs bvh culling time "<<timer.ElapsedMillis()<<" ms"<<std::endl;
	h_list = d_vector1;
	k = 0;
	for(int i=0; i<size*frustumCount; i++)
		if(h_list[i] == 1)
			k++;
	std::cout<<"total "<<h_list.size()<<" in "<<k<<std::endl;
	hipFree(d_nbvh);
	delete[] nbvh;

	Bintree_node * Nbvh;
	Nbvh = new Bintree_node[nbvh_size];   
	ctimer.start();
	cuda::BFSBintree(&h_nodes,&h_leaves,&h_nodeBoxes,&h_leafBoxes,Nbvh);
	ctimer.stop();
	std::cout<<"build BFSBintree time "<<ctimer.seconds()*1000<<"ms"<<std::endl;
	Bintree_node* d_Nbvh;
	hipMalloc((void**)&d_Nbvh,sizeof(Bintree_node)*nbvh_size);
	hipMemcpy(d_Nbvh,Nbvh,sizeof(Bintree_node)*nbvh_size,hipMemcpyHostToDevice);
	thrust::device_vector<uint32> d_vector0(size*frustumCount,0);
	d_list = thrust::raw_pointer_cast(&d_vector0.front());
	timer.Start();
	FrustumCullingKernel<<<n_blocks*numSMs,BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_frustums.front()),frustumCount, d_Nbvh, size,d_list);
	timer.Stop();
	std::cout<<"BFS Nbvh culling time "<<timer.ElapsedMillis()<<" ms"<<std::endl;
	h_list = d_vector0;
	k = 0;
	for(int i=0; i<size*frustumCount; i++)
		if(h_list[i] == 1)
			k++;
	std::cout<<"total "<<h_list.size()<<" in "<<k<<std::endl;
	hipFree(d_Nbvh);
	delete[] Nbvh;



	//Bintree h_bvhSoa;
	//cuda::DFSBintreeSOA(&h_nodes,&h_leaves,&h_nodeBoxes,&h_leafBoxes,&h_bvhSoa);
	//Bintree* d_bvhSoa;
	//hipMalloc((void**)&d_bvhSoa,sizeof(Bintree));
	//hipMemcpy(d_bvhSoa,&h_bvhSoa,sizeof(Bintree),hipMemcpyHostToDevice);
	//thrust::device_vector<uint32> d_vector3(size*frustumCount,0);
	//d_list = thrust::raw_pointer_cast(&d_vector3.front());
	//timer.Start();
	//FrustumCullingKernel<<<n_blocks*numSMs,BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_frustums.front()),frustumCount, d_bvhSoa, size,d_list);
	//timer.Stop();
	//std::cout<<"dfs bvh soa culling time "<<timer.ElapsedMillis()<<" ms"<<std::endl;
	//h_list = d_vector3;
	//k = 0;
	//for(int i=0; i<size*frustumCount; i++)
	//	if(h_list[i] == 1)
	//		k++;
	//std::cout<<"total "<<h_list.size()<<" in "<<k<<std::endl;
	//hipFree(h_bvhSoa.boxPtr);
	//hipFree(h_bvhSoa.isLeafPtr);
	//hipFree(h_bvhSoa.LChildPtr);
	//hipFree(h_bvhSoa.leafEndPtr);
	//hipFree(h_bvhSoa.leafStartPtr);
	//hipFree(h_bvhSoa.pidPtr);
	//hipFree(h_bvhSoa.RChildPtr);
	//hipFree(d_bvhSoa);

	thrust::device_vector<uint32> d_vector2(size,0);
	d_list = thrust::raw_pointer_cast(&d_vector2.front());
	n_blocks   = nih::min( max_blocks, (size + (BLOCK_SIZE*numSMs)-1) / (BLOCK_SIZE*numSMs) );
	timer.Start();
	BruteforceFrustumCullingKernel<<<n_blocks*numSMs,BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_frustums.front()), builder.getLeafBoxes(), size,d_list);
	timer.Stop();
	std::cout<<"brute force culling time "<<timer.ElapsedMillis()<<" ms"<<std::endl;
	h_list = d_vector2;
	k = 0;
	for(int i=0; i<size; i++)
		if(h_list[i] == 1)
			k++;
	std::cout<<"total "<<h_list.size()<<" in "<<k<<std::endl;




	/*cuda::SimpleList<Pair> h_plist(h_points.size()*2);
	cuda::SimpleList<Pair>* d_plist=NULL;
	hipMalloc((void**)(&d_plist),sizeof(cuda::SimpleList<Pair>));
	hipMemcpy(d_plist,&h_plist,sizeof(cuda::SimpleList<Pair>),hipMemcpyHostToDevice);
	const size_t blocks   = nih::min( max_blocks, (size + (BLOCK_SIZE*numSMs)-1) / (BLOCK_SIZE*numSMs) );
	timer.Start();
	CollisonDetectionKernel<<<blocks*numSMs,BLOCK_SIZE>>>(d_bvh,d_plist,size);
	timer.Stop();
	std::cout<<"collison detec time "<<timer.ElapsedMillis()<<" ms"<<std::endl;
	hipMemcpy(&h_plist,d_plist,sizeof(cuda::SimpleList<uint32>),hipMemcpyDeviceToHost);
	std::cout<<"overlapped "<<h_plist.size()<<std::endl;
	*/
	//hipFree(d_bvh);
}

bool BboxCompare(const Bbox3f& lbox, const Bbox3f& rbox)
{
	const double zero = 0.0001;

	return (abs(lbox.m_min[0]-rbox.m_min[0])<zero &&
		abs(lbox.m_min[1]-rbox.m_min[1])<zero && 
		abs(lbox.m_min[2]-rbox.m_min[2])<zero &&
		abs(lbox.m_max[0]-rbox.m_max[0])<zero &&
		abs(lbox.m_max[1]-rbox.m_max[1])<zero && 
		abs(lbox.m_max[2]-rbox.m_max[2])<zero );
}
struct is_frustum
{
	NIH_HOST_DEVICE bool  operator()(const TriFrustum& f)
	{
		return f.id != uint32(-1);
	}
};
struct add_frustum
{
	NIH_HOST_DEVICE TriFrustum operator()(const TriFrustum& a, const TriFrustum& b)
	{
		Bbox3f aBox(a.min,a.max);
		Bbox3f bBox(b.min,b.max);
		aBox.insert(bBox);
		TriFrustum ret;
		ret.max = aBox.m_max;
		ret.min = aBox.m_min;
		return ret;
	}
};
void PlaneIntersectTest()
{
	plane_t plane;
	plane.a = 0.21;
	plane.b = 0.04;
	plane.c = -0.97;
	plane.d = -4.4;
	Vector3f p0(4.95,0.99,0.25);
	Vector3f p1(0.18,0.02,-0.98);
	Vector3f p;
	plane.intersect(p0,p1,p);
}

void BuildSceneBVH(thrust::host_vector<Vector3f>& sceneCenters,
	thrust::host_vector<Bbox3f>& sceneBoxes, Bbox3f& scnGBox)
{
	size_t bvhSize = sceneCenters.size();
	thrust::device_vector<Bvh_Node> nodes(bvhSize-1);
	thrust::device_vector<Bvh_Node> leaves(bvhSize);
	hipcub::CachingDeviceAllocator allocator(true);
	cuda::KBvh_Builder builder(nodes,leaves,allocator);

	thrust::device_vector<Bbox3f> d_boxes(sceneBoxes);
	thrust::device_vector<Vector3f> d_centers(sceneCenters);
	cuda::DBVH h_bvh;
	builder.build(scnGBox,d_centers.begin(),d_centers.end(),d_boxes.begin(),d_boxes.end(),&h_bvh);

	thrust::host_vector<Bvh_Node> h_nodes(nodes);
	thrust::host_vector<Bvh_Node> h_leaves(leaves);	
	thrust::host_vector<Bbox3f> h_nodeBoxes(bvhSize-1);
	thrust::host_vector<Bbox3f> h_leafBoxes(bvhSize);


	//for(int i = 0; i<h_nodes.size(); i++)
	//{ 
	//	std::cout<<" parent idx is "<<h_nodes[i].parentIdx<<" ,";

	//	if(h_nodes[i].l_isleaf)
	//	{
	//		std::cout<<i<<" left child "<<" is leaf "<<h_nodes[i].getChild(0);
	//	}
	//	else
	//	{
	//		std::cout<<i<<" left child "<<" is internal "<<h_nodes[i].getChild(0);				

	//	}
	//	if(h_nodes[i].r_isleaf)
	//	{
	//		std::cout<<" right child "<<" is leaf "<<h_nodes[i].getChild(1)<<std::endl;
	//	}
	//	else
	//	{
	//		std::cout<<" right child "<<" is internal "<<h_nodes[i].getChild(1)<<std::endl;
	//	}
	//}
	//for(int i=0; i<h_leaves.size(); i++)
	//{
	//	std::cout<<i<<" parent is "<<h_leaves[i].parentIdx<<std::endl;
	//	std::cout<<" pid is "<<h_leaves[i].pid<<std::endl;
	//}


	Bbox3f* p_nodeBoxes = thrust::raw_pointer_cast(&h_nodeBoxes.front());
	hipMemcpy(p_nodeBoxes,builder.getNodeBoxes(),sizeof(Bbox3f)*(bvhSize-1),hipMemcpyDeviceToHost);


	Bbox3f* p_leafBoxes = thrust::raw_pointer_cast(&h_leafBoxes.front());
	hipMemcpy(p_leafBoxes,builder.getLeafBoxes(),sizeof(Bbox3f)*(bvhSize),hipMemcpyDeviceToHost);
	//ת��ΪDFS���е���
	Bintree_node* nbvh;
	uint32 nbvh_size = bvhSize*2-1;
	nbvh = new Bintree_node[nbvh_size];	
	cuda::DFSBintree(&h_nodes,&h_leaves,&h_nodeBoxes,&h_leafBoxes,nbvh);		


	hipMalloc((void**)&gd_nbvh,sizeof(Bintree_node)*nbvh_size);
	hipMemcpy(gd_nbvh,nbvh,sizeof(Bintree_node)*nbvh_size,hipMemcpyHostToDevice);
	hipBindTexture( NULL, bvhTex,
		gd_nbvh, sizeof(Bintree_node)*nbvh_size );

	hipMalloc((void**)&gd_indices,sizeof(uint32)*bvhSize);
	hipMemcpy(gd_indices,builder.getIndices(),sizeof(uint32)*bvhSize,hipMemcpyDeviceToDevice);
	hipBindTexture(NULL,indexTex,gd_indices,sizeof(uint32)*bvhSize);


	delete[] nbvh;
	//std::cout<<"bvh buided"<<std::endl;
}
template<typename T,typename SelOp>
int SelectIf(T* input, T* output, int num_items, SelOp& op)
{	
	int      *d_num_selected;    
	hipMalloc((void**)&d_num_selected,sizeof(int));
	void     *d_temp_storage = NULL;
	size_t   temp_storage_bytes = 0;

	hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, input, output, d_num_selected, num_items, op);
	// Allocate temporary storage
	hipMalloc(&d_temp_storage, temp_storage_bytes);
	// Run selection
	hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, input, output, d_num_selected, num_items,op);
	int h_selected;
	hipMemcpy(&h_selected,d_num_selected,sizeof(int),hipMemcpyDeviceToHost);		
	hipFree(d_temp_storage);
	hipFree(d_num_selected);
	return h_selected;
}

void NaiveCulling(TriFrustum* d_frustums, uint32 frustumNum, thrust::host_vector<Vector3f>& sceneCenters,
	thrust::host_vector<Bbox3f>& sceneBoxes, Bbox3f& scnGBox)
{
	GpuTimer timer;
	timer.Start();

	//����BVH
	BuildSceneBVH(sceneCenters,sceneBoxes,scnGBox);
	size_t nbvh_size = sceneCenters.size();
	//�ü�
	size_t cullingSize = frustumNum* nbvh_size;

	thrust::device_vector<cullingContext> d_listVec(cullingSize);
	cullingContext* d_list = thrust::raw_pointer_cast(&d_listVec.front());
	size_t n_blocks = GridSize(frustumNum);		
	FrustumCullingKernel<<<n_blocks,128>>>(d_frustums,frustumNum, nbvh_size,d_list);

	cullingContext* cullingResult;
	hipMalloc(&cullingResult,sizeof(cullingContext)*cullingSize);
	int h_num_selected = SelectIf(d_list,cullingResult,cullingSize,is_valid());
	timer.Stop();
	std::cout<<"NaiveCulling total "<<cullingSize<<" in "<<h_num_selected<<"  "<<timer.ElapsedMillis()<<" ms "<<std::endl;

	hipFree(cullingResult);
	//�ͷ�bvh
	hipFree(gd_nbvh);
	hipFree(gd_indices);
}

void BruteForceCulling(TriFrustum* d_frustums,size_t frustumNum, thrust::host_vector<Bbox3f>& h_boxes)
{
	GpuTimer timer;
	timer.Start();
	thrust::device_vector<Bbox3f> d_boxesVec(h_boxes);
	Bbox3f* d_boxes = thrust::raw_pointer_cast(&d_boxesVec.front());
	size_t size = h_boxes.size();
	size_t gridSize = GridSize(size);
	thrust::device_vector<cullingContext> d_clist(size*frustumNum);
	thrust::device_vector<cullingContext> d_flist(size*frustumNum);

	cullingContext* d_listIn = thrust::raw_pointer_cast(&d_clist.front());
	cullingContext* d_listOut = thrust::raw_pointer_cast(&d_flist.front());
	BruteforceFrustumCullingKernel<<<gridSize,128>>>(d_frustums,frustumNum,d_boxes,size,d_listIn);

	int inNum = SelectIf(d_listIn,d_listOut,size*frustumNum,is_valid());
	timer.Stop();

	std::cout<<"bruteforce culling  in "<<inNum<<" "<<timer.ElapsedMillis()<<"ms"<<std::endl;
}

void SmartCulling(TriFrustum* d_frustums, uint32 frustumNum, thrust::host_vector<Vector3f>& sceneCenters,
	thrust::host_vector<Bbox3f>& sceneBoxes, Bbox3f& scnGBox)
{
	//��׶����morton code����
	//����������׶���ܰ�Χ��
	thrust::device_ptr<TriFrustum> dptr(d_frustums);
	TriFrustum gFrustum = thrust::reduce(dptr,dptr+frustumNum,TriFrustum(),add_frustum());
	Bbox3f frustumBox(gFrustum.min,gFrustum.max);
	thrust::device_vector<Bbox3f> d_frustumBoxs(1,frustumBox);
	Bbox3f* d_frustumBox = thrust::raw_pointer_cast(&d_frustumBoxs.front());
	thrust::device_vector<Vector3f> d_pointsVec(frustumNum);
	thrust::device_vector<Bbox3f> d_boxesVec(frustumNum);
	Vector3f* d_points = thrust::raw_pointer_cast(&d_pointsVec.front());
	Bbox3f* d_boxes = thrust::raw_pointer_cast(&d_boxesVec.front());

	size_t gridSize = GridSize(frustumNum);
	PrepareBvhPointKernel<<<gridSize,128>>>(d_frustums,d_points,d_boxes,frustumNum);
	hipcub::DoubleBuffer<uint32> d_codes;
	hipcub::CachingDeviceAllocator  allocator(true);
	CubDebugExit(allocator.DeviceAllocate((void**)&d_codes.d_buffers[0], sizeof(uint32) * frustumNum));
	CubDebugExit(allocator.DeviceAllocate((void**)&d_codes.d_buffers[1], sizeof(uint32) * frustumNum));
	hipcub::DoubleBuffer<uint32> d_indices;
	CubDebugExit(allocator.DeviceAllocate((void**)&d_indices.d_buffers[0], sizeof(uint32) * frustumNum));
	CubDebugExit(allocator.DeviceAllocate((void**)&d_indices.d_buffers[1], sizeof(uint32) * frustumNum));
	hipcub::DoubleBuffer<TriFrustum> d_frustumBuffer;
	d_frustumBuffer.d_buffers[0] = d_frustums;
	CubDebugExit(allocator.DeviceAllocate((void**)&d_frustumBuffer.d_buffers[1], sizeof(TriFrustum) * frustumNum));

	//����morton code
	thrust::transform(
		d_pointsVec.begin(),
		d_pointsVec.end(),
		thrust::device_ptr<uint32>(d_codes.d_buffers[0]),
		morton_functor<uint32>( frustumBox ) );
	/*Allocate temporary storage*/
	uint32 temp_storage_bytes  = 0;
	void* d_temp_storage     = NULL;
	CubDebugExit(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_codes, d_frustumBuffer, frustumNum));
	CubDebugExit(allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
	//����
	thrust::copy(
		thrust::counting_iterator<uint32>(0),
		thrust::counting_iterator<uint32>(0) + frustumNum,
		thrust::device_ptr<uint32>(d_indices.d_buffers[0]) );
	CubDebugExit(hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_codes, d_frustumBuffer, frustumNum));

	/*thrust::host_vector<uint32> h_codes(frustumNum);
	uint32* h_codesPtr = thrust::raw_pointer_cast(&h_codes.front());

	hipMemcpy(h_codesPtr,d_codes.d_buffers[1],sizeof(uint32)*frustumNum,hipMemcpyDeviceToHost);
	for(int i=0; i<frustumNum; i++)
	{
	std::cout<<h_codes[i]<<std::endl;
	}*/

	NaiveCulling(d_frustumBuffer.d_buffers[1],frustumNum,sceneCenters,sceneBoxes,scnGBox);
	allocator.DeviceFree(d_temp_storage);
	allocator.DeviceFree(d_indices.d_buffers[0]);
	allocator.DeviceFree(d_indices.d_buffers[1]);
	allocator.DeviceFree(d_codes.d_buffers[0]);
	allocator.DeviceFree(d_codes.d_buffers[1]);
	allocator.DeviceFree(d_frustumBuffer.d_buffers[1]);


}
void SmartCulling_Err(TriFrustum* d_frustums, uint32 frustumNum, thrust::host_vector<Vector3f>& h_points)
{
	GpuTimer timer;
	timer.Start();
	//����������׶���ܰ�Χ��
	thrust::device_ptr<TriFrustum> dptr(d_frustums);
	TriFrustum gFrustum = thrust::reduce(dptr,dptr+frustumNum,TriFrustum(),add_frustum());
	Bbox3f frustumBox(gFrustum.min,gFrustum.max);
	thrust::device_vector<Bbox3f> d_frustumBoxs(1,frustumBox);
	Bbox3f* d_frustumBox = thrust::raw_pointer_cast(&d_frustumBoxs.front());

	size_t size = h_points.size();
	thrust::device_vector<Vector3f> d_ScenePointVec(h_points);
	thrust::device_vector<Vector3f> d_SelScnPointsVec(h_points.size());
	Vector3f* d_ScenePoints = thrust::raw_pointer_cast(&d_ScenePointVec.front());
	Vector3f* d_SelScnPoints = thrust::raw_pointer_cast(&d_SelScnPointsVec.front());
	thrust::device_vector<char> d_vector2(size,0);
	char* d_list = thrust::raw_pointer_cast(&d_vector2.front());
	size_t gridSize = GridSize(size/3);

	BboxCullingKernel<<<gridSize,128>>>(d_frustumBox, d_ScenePoints, size/3,d_list);
	//timer.Stop();
	//std::cout<<"brute force culling time "<<timer.ElapsedMillis()<<" ms"<<std::endl;

	//timer.Start();
	int num_items = size;
	int * d_num_selected;
	hipMalloc(&d_num_selected,sizeof(int));
	// Determine temporary device storage requirements	
	size_t temp_storage_bytes = 0;
	void* d_temp_storage = NULL;
	hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_ScenePoints, d_list, d_SelScnPoints, d_num_selected, num_items);
	// Allocate temporary storage
	hipMalloc(&d_temp_storage, temp_storage_bytes);
	// Run selection
	int h_num_selected;
	hipcub::DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_ScenePoints, d_list, d_SelScnPoints, d_num_selected, num_items);
	hipMemcpy(&h_num_selected,d_num_selected,sizeof(int),hipMemcpyDeviceToHost);
	//std::cout<<"culling with gfrustum box, remaining "<<h_num_selected<<std::endl;
	hipFree(d_num_selected);
	hipFree(d_temp_storage);
	//timer.Stop();
	//std::cout<<"stream compaction time "<<timer.ElapsedMillis()<<" ms"<<std::endl;


	//����bvh
	uint32 size0 = h_num_selected/3;
	size_t bvhSize = size0 +frustumNum;	
	thrust::device_vector<Vector3f> d_bvhPoints(bvhSize);
	thrust::device_vector<Bbox3f> d_bvhBoxes(bvhSize );
	Vector3f* d_points0 = thrust::raw_pointer_cast(&d_bvhPoints.front());
	Vector3f* d_points1 = thrust::raw_pointer_cast(&d_bvhPoints[size0]);
	Bbox3f* d_boxes0 = thrust::raw_pointer_cast(&d_bvhBoxes.front());
	Bbox3f* d_boxes1 = thrust::raw_pointer_cast(&d_bvhBoxes[size0]);
	gridSize = GridSize(size0);
	PrepareBvhPointKernel<<<gridSize,128>>>(d_SelScnPoints,d_points0,d_boxes0,size0);
	gridSize = GridSize(frustumNum);
	PrepareBvhPointKernel<<<gridSize,128>>>(d_frustums,d_points1,d_boxes1,frustumNum);
	nih::Bbox3f h_gBox = thrust::reduce(d_bvhBoxes.begin(),d_bvhBoxes.end(),nih::Bbox3f(),Add_Bbox<nih::Vector3f>());

	thrust::device_vector<Bvh_Node> nodes(bvhSize-1);
	thrust::device_vector<Bvh_Node> leaves(bvhSize);
	hipcub::CachingDeviceAllocator allocator(true);
	cuda::KBvh_Builder builder(nodes,leaves,allocator);


	cuda::DBVH h_bvh;


	//timer.Start();
	builder.build(h_gBox,d_bvhPoints.begin(),d_bvhPoints.end(),d_bvhBoxes.begin(),d_bvhBoxes.end(),&h_bvh);
	//timer.Stop();
	//std::cout<<"build time "<<timer.ElapsedMillis()<<" ms"<<std::endl;

	thrust::host_vector<Bvh_Node> h_nodes = nodes;
	thrust::host_vector<Bvh_Node> h_leaves = leaves;
	for(int i = 0; i<h_nodes.size(); i++)
	{ 
		std::cout<<" parent idx is "<<h_nodes[i].parentIdx<<" ,";

		if(h_nodes[i].l_isleaf)
		{
			std::cout<<i<<" left child "<<" is leaf "<<h_nodes[i].getChild(0);
		}
		else
		{
			std::cout<<i<<" left child "<<" is internal "<<h_nodes[i].getChild(0);				

		}
		if(h_nodes[i].r_isleaf)
		{
			std::cout<<" right child "<<" is leaf "<<h_nodes[i].getChild(1)<<std::endl;
		}
		else
		{
			std::cout<<" right child "<<" is internal "<<h_nodes[i].getChild(1)<<std::endl;
		}
	}
	for(int i=0; i<h_leaves.size(); i++)
	{
		std::cout<<i<<" parent is "<<h_leaves[i].parentIdx<<std::endl;
		std::cout<<" pid is "<<h_leaves[i].pid<<std::endl;
	}
	//timer.Start();
	cuda::DBVH* d_bvh = NULL;
	hipMalloc((void**)&d_bvh,sizeof(BVH));
	hipMemcpy(d_bvh,&h_bvh,sizeof(BVH),hipMemcpyHostToDevice);

	gridSize = GridSize(bvhSize);
	size_t totalSize = frustumNum*bvhSize;
	thrust::device_vector<cullingContext> d_cullingResult(totalSize);
	cullingContext* d_clist = thrust::raw_pointer_cast(&d_cullingResult.front());
	//FrustumCullingKernel<<<gridSize,128>>>(d_frustumsOut,frustumNum,d_bvh, size0,d_clist);
	FrustumCullingKernel<<<gridSize,128>>>(d_frustums,d_bvh,frustumNum, size0,d_clist);

	thrust::device_vector<cullingContext> d_fcullingResult(totalSize);
	cullingContext* d_fclist = thrust::raw_pointer_cast(&d_fcullingResult.front());
	int selectedNum = SelectIf(d_clist,d_fclist,totalSize,is_valid());

	timer.Stop();
	std::cout<<"smart culling total "<<totalSize<<" in "<<selectedNum<<" "<<timer.ElapsedMillis()<<" ms"<<std::endl;

}
void TestSelectIf()
{
	const size_t cullingSize = 4518600;
	cullingContext* h_list = new cullingContext[cullingSize];
	for(int i=0; i<6; i++)
		h_list[i].triId = i;
	cullingContext* cullingResult;
	hipMalloc((void**)&cullingResult,cullingSize*sizeof(cullingContext));
	cullingContext* cullingInput;
	hipMalloc((void**)&cullingInput,cullingSize*sizeof(cullingContext));
	hipMemcpy(cullingInput,h_list,sizeof(cullingContext)*cullingSize,hipMemcpyHostToDevice);

	int ret = SelectIf(cullingInput,cullingResult,cullingSize,is_valid());
	std::cout<<ret<<std::endl;
	hipFree(cullingResult);
	hipFree(cullingInput);
	delete[] h_list;
}
void GPUVirtualFrustumTest()
{
	const char* mirrorObjName = "sphere4900.obj";
	const char* sceneObjName = "sponza.obj";
	thrust::host_vector<Vector3f> h_p123Vec;	
	size_t mirrorTriNum = loadObj(mirrorObjName,h_p123Vec);

	thrust::host_vector<Vector3f> h_centers;
	thrust::host_vector<Bbox3f> h_boxes;
	Bbox3f gBox;
	loadObj(sceneObjName,h_centers,h_boxes,gBox);


	thrust::device_vector<Vector3f> d_p123Vec(h_p123Vec);
	Vector3f* d_p123 = thrust::raw_pointer_cast(&d_p123Vec.front());

	Vector3f eye(4,1,0);
	thrust::device_vector<Vector3f> d_eyeVec(1,eye);
	Vector3f* d_eye = thrust::raw_pointer_cast(&d_eyeVec.front());

	thrust::device_vector<TriFrustum> d_frustumVec(mirrorTriNum);
	TriFrustum* d_frustums = thrust::raw_pointer_cast(&d_frustumVec.front());

	size_t gridSize = GridSize(mirrorTriNum);

	GenerateVirFrustumKernel<<<gridSize,128>>>(d_eye,d_p123, d_frustums,150, mirrorTriNum);

	int      num_items = mirrorTriNum;     
	TriFrustum      *d_frustumsOut;             
	hipMalloc((void**)&d_frustumsOut,sizeof(TriFrustum)*mirrorTriNum);
	int frustumNum = SelectIf(d_frustums,d_frustumsOut,num_items,is_frustum());
	//int      *d_num_selected;    
	//hipMalloc((void**)&d_num_selected,sizeof(int));
	//void     *d_temp_storage = NULL;
	//size_t   temp_storage_bytes = 0;
	//is_frustum op;
	//hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_frustums, d_frustumsOut, d_num_selected, num_items, op);
	//// Allocate temporary storage
	//hipMalloc(&d_temp_storage, temp_storage_bytes);
	//// Run selection
	//hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_frustums, d_frustumsOut, d_num_selected, num_items,op);
	//int frustumNum;
	//hipMemcpy(&frustumNum,d_num_selected,sizeof(int),hipMemcpyDeviceToHost);		
	//hipFree(d_temp_storage);
	//std::cout<<"frustum generated "<<frustumNum<<std::endl;


	BruteForceCulling(d_frustumsOut,frustumNum,h_boxes);
	NaiveCulling(d_frustumsOut,frustumNum,h_centers,h_boxes,gBox);

	thrust::host_vector<Vector3f> h_points;
	size_t size = loadObj(sceneObjName,h_points);
	SmartCulling(d_frustumsOut,frustumNum,h_centers,h_boxes,gBox);

	hipFree(d_frustumsOut);
}
int main(int argc, char** argv)
{

	char* fileName ="testbox.obj";
	if (argc == 2)
	{
		fileName = (argv[1]);
	}

	uint32 tableXX[2] = {3,0};
	uint32 tableYY[2] = {4,1};
	uint32 tableZZ[2] = {5,2};
	uint32 tableX[2] = {0,3};
	uint32 tableY[2] = {1,4};
	uint32 tableZ[2] = {2,5};
	hipMemcpyToSymbol(HIP_SYMBOL( tx),  tableX,   sizeof(uint32)*2  );
	hipMemcpyToSymbol(HIP_SYMBOL( ty),  tableY,   sizeof(uint32)*2  );
	hipMemcpyToSymbol(HIP_SYMBOL( tz),  tableZ,   sizeof(uint32)*2  );
	hipMemcpyToSymbol(HIP_SYMBOL( ttx),  tableXX,   sizeof(uint32)*2  );
	hipMemcpyToSymbol(HIP_SYMBOL( tty),  tableYY,   sizeof(uint32)*2  );
	hipMemcpyToSymbol(HIP_SYMBOL( ttz),  tableZZ,   sizeof(uint32)*2  );

	GPUVirtualFrustumTest();
	return;

	// Projection matrix : 45�� Field of View, 4:3 ratio, display range : 0.1 unit <-> 100 units
	glm::mat4 Projection = glm::perspective(45.0f, 4.0f / 3.0f, 0.01f, 50.0f);
	// Camera matrix
	glm::mat4 View       = glm::lookAt(
		glm::vec3(0,0,-7), // Camera is at (4,3,3), in World Space
		glm::vec3(0,0,0), // and looks at the origin
		glm::vec3(0,1,0)  // Head is up (set to 0,-1,0 to look upside-down)
		);
	// Model matrix : an identity matrix (model will be at the origin)
	glm::mat4 Model      = glm::mat4(1.0f);  // Changes for each model !

	// Our ModelViewProjection : multiplication of our 3 matrices
	glm::mat4 MVP        = Projection * View * Model; // Remember, matrix multiplication is the other way around

	Matrix4x4 mvp;		
	memcpy(&mvp,&MVP[0][0],16*sizeof(float));
	pyrfrustum_t frustum;
	ExtractPlanesGL(frustum.planes,mvp,true);

	thrust::host_vector<Vector3f> h_points;
	thrust::host_vector<Bbox3f> h_boxes;
	Bbox3f gBox;
	loadObj(fileName,h_points,h_boxes,gBox);
	//loadRandom(3650,h_points,h_boxes,gBox);

	thrust::host_vector<Bvh_Node> h_nodes,h_leaves;
	thrust::host_vector<Bbox3f> h_nodeBoxes,h_leafBoxes;
	std::cout<<"gpu:"<<std::endl;
	GpuKlbvhTest(h_points,h_boxes,gBox,frustum,h_nodes,h_leaves,h_nodeBoxes,h_leafBoxes);
	std::cout<<"cpu:"<<std::endl;
	BVH cpuBvh;
	CpuKlbvhTest(h_points,h_boxes,gBox,frustum,cpuBvh);

	std::cout<<"�Ƚ�"<<std::endl;
	if (cpuBvh.nodes.size() != h_nodes.size())
	{
		std::cout<<"size is different!"<<std::endl;
	}
	for (int i=0; i<cpuBvh.nodes.size(); i++)
	{
		if (!cpuBvh.nodes[i].equal(h_nodes[i]))
		{
			std::cout<<"node "<<i<<"is different"<<std::endl;
			break;
		}
		if (cpuBvh.leafs[i].parentIdx != h_leaves[i].parentIdx)
		{
			std::cout<<"leaf "<<i<<"is different"<<std::endl;
			std::cout<<"cpu "<<cpuBvh.leafs[i].parentIdx<<" gpu "<<h_leaves[i].parentIdx<<std::endl;
			break;
		}
		if (!BboxCompare(cpuBvh.node_Boxes[i],h_nodeBoxes[i]))
		{
			std::cout<<"node box "<<i<<"is different"<<std::endl;
			break;
		}
		if (!BboxCompare(cpuBvh.leaf_Boxes[i],h_leafBoxes[i]))
		{
			std::cout<<"leaf box "<<i<<"is different"<<std::endl;
			std::cout<<"cpu ";
			printBbox3f(cpuBvh.leaf_Boxes[i]);
			std::cout<<"gpu ";
			printBbox3f(h_leafBoxes[i]);
			break;
		}
	}

	return 0;
}
