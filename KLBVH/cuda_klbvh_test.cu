#include "hip/hip_runtime.h"
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include "cuda_klbvh.h"
#include "MortonCode.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "objLoader.h"
#include <algorithm>
#include <cstdlib>
#include "klbvh.h"
#include "gputimer.cuh"
#include "timer.h"
using namespace nih;
int globalCounter = 0;
texture<float> BboxTex;
texture<uint32> bvhTex;
struct bvhTexHelper
{
	static const uint32 nodeSize = 11;
	static const uint32 LChildOf = 6;
	static const uint32 RChildOf = 7;
	static const uint32 pidOf = 8;
	static const uint32 leafStartOf = 9;
	static const uint32 leafEndOf = 10;
	float p[6];
	NIH_DEVICE float* getBbox(uint32 id)
	{
		uint32 offset = id*nodeSize;
		uint32 t[6];
		
		for(int i=0; i<6; i++)
		{
			t[i] = tex1Dfetch(bvhTex,offset+i);
			p[i] = bitsToFloat(t[i]);
		}

	
		return p;
	}

	NIH_DEVICE uint32 getLChild(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+LChildOf);
	}

	NIH_DEVICE uint32 getRChild(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+RChildOf);
	}
	NIH_DEVICE uint32 getPid(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+pidOf);
	}
	NIH_DEVICE uint32 getleafStart(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+leafStartOf);
	}
	NIH_DEVICE uint32 getleafEnd(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+leafEndOf);
	}
	NIH_DEVICE bool isLeaf(uint32 id)
	{
		return getleafStart(id) == getleafEnd(id);
	}
};
NIH_DEVICE bool AABBOverlap(Bbox3f& boxA, Bbox3f& boxB)
{
	for (int i=0; i<3; i++)
	{
		if (fabs(boxB.m_max[i]+boxB.m_min[i]-boxA.m_max[i]-boxA.m_min[i]) <
			boxA.m_max[i]-boxA.m_min[i] + boxB.m_max[i] - boxB.m_min[i])
			return true;
	}
	return false;
}
//FORCE_INLINE NIH_DEVICE void traverseIterative(Bbox3f& qbox, uint32 qId,uint32 leafIdx,
//	cuda::DBVH* bvh,
//	uint32* list)
//{
//	Bvh_Node* stack[64];
//	uint32 top = 0;
//	stack[top++] = bvh->getRoot();
//	while(top>0)
//	{
//		Bvh_Node* node = stack[--top];
//		if (node->leafEnd <= leafIdx)
//			continue;
//		Bbox3f box = bvh->getNodeBox(node);
//		
//		bool ret = AABBOverlap(qbox, box);
//		if (ret )
//		{
//			//�ཻ
//			if (node->l_isleaf)
//			{
//				Bvh_Node* leaf = (bvh->getLLeafChild(node));
//				Bbox3f box = bvh->getLeafBox(leaf);
//				if (AABBOverlap(qbox,box))
//				{
//					list->add(Pair(qId,leaf->pid));
//				}				
//			}
//			else
//				stack[top++] = bvh->getLChild(node);
//			if(node->r_isleaf)
//			{
//				Bvh_Node* leaf = (bvh->getRLeafChild(node));
//				Bbox3f box = bvh->getLeafBox(leaf);
//				if (AABBOverlap(qbox,box))
//				{
//					list->add(Pair(qId,leaf->pid));
//				}				
//			}
//			else
//				stack[top++] =  bvh->getRChild(node);
//		}		
//	}
//}
FORCE_INLINE NIH_DEVICE void FrustumCulling(pyrfrustum_t& frustum, uint32 frustumId,
	cuda::DBVH* bvh,uint32 priSize,
	uint32* list)
{
	Bvh_Node* stack[64];
	uint32 top = 0;
	stack[top++] = bvh->getRoot();
	uint32 offset = priSize*frustumId;
	while(top>0)
	{
		Bvh_Node* node = stack[--top];
		Bbox3f box = bvh->getNodeBox(node);
		
		int ret = Intersect(frustum, box);
		if (ret == 2)
		{
			//�ཻ
			if (node->l_isleaf)
			{
				Bvh_Node* leaf = (bvh->getLLeafChild(node));
				Bbox3f box = bvh->getLeafBox(leaf);
				if (Intersect(frustum,box))
				{
					list[offset+leaf->pid] = 1;
				}				
			}
			else
				stack[top++] = bvh->getLChild(node);
			if(node->r_isleaf)
			{
				Bvh_Node* leaf = (bvh->getRLeafChild(node));
				Bbox3f box = bvh->getLeafBox(leaf);
				if (Intersect(frustum,box))
				{
					list[offset+leaf->pid] = 1;
				}				
			}
			else
				stack[top++] =  bvh->getRChild(node);
		}
		else if (ret == 1)
		{
			//in
			for(int k= node->leafStart; k<=node->leafEnd;k++)
				list[offset+bvh->leaves[k].pid] = 1;
		}
	}
}
FORCE_INLINE NIH_DEVICE void FrustumCulling(pyrfrustum_t& frustum, uint32 frustumId,
	Bintree* bvh,uint32 priSize,
	uint32* out)
{
	uint32 offset = priSize*frustumId;
	const uint32 stack_size  = 64;
	uint32 stack[stack_size];
	uint32 top = 0;
	stack[top++] = 0;
	while(top>0)
	{
		uint32 idx = stack[--top];		
		int ret = Intersect(frustum,bvh->boxPtr[idx]);
		if (ret == 2)
		{
			//�ཻ
			
			if(bvh->isLeafPtr[bvh->RChildPtr[idx]])
			{
				if (Intersect(frustum,bvh->boxPtr[bvh->RChildPtr[idx]]))
				{
					out[offset+bvh->pidPtr[bvh->RChildPtr[idx]]] = 1;					
				}				
			}
			else
				stack[top++] = bvh->RChildPtr[idx];

			if (bvh->isLeafPtr[bvh->LChildPtr[idx]])
			{
				
				if (Intersect(frustum,bvh->boxPtr[bvh->LChildPtr[idx]]))
				{
					out[offset+bvh->pidPtr[bvh->LChildPtr[idx]]] = 1;					
				}				
			}
			else
				stack[top++] = bvh->LChildPtr[idx];
		}
		else if (ret == 1)
		{
			//in
			for(int k= bvh->leafStartPtr[idx]; k<=bvh->leafEndPtr[idx];k++)
				out[offset+offset+bvh->pidPtr[k]] = 1;
			
		}
	}
}
FORCE_INLINE NIH_DEVICE void FrustumCulling(pyrfrustum_t& frustum, uint32 frustumId,
	Bintree_Node* bvh,uint32 priSize,
	uint32* out)
{
	uint32 offset = priSize*frustumId;
	const uint32 stack_size  = 64;
	Bintree_Node* stack[stack_size];
	Bintree_Node** stackPtr = stack;
	*stackPtr++ = NULL;
	Bintree_Node* node = &bvh[0];
	do
    {
        // Check each child node for overlap.
		Bintree_Node* childL = &bvh[node->lChild];
		Bintree_Node* childR = &bvh[node->RChild];
        int overlapL = ( Intersect(frustum, 
			node->lBox) );
        int overlapR = ( Intersect(frustum, 
			node->rBox) );

        // Query overlaps a leaf node => report collision.
		if (overlapL && bvh[node->lChild].isLeaf())
			out[offset + bvh[node->lChild].leafStart] = 1;

		if (overlapR && bvh[node->RChild].isLeaf())
            out[offset + bvh[node->RChild].leafStart] = 1;
		
        // Query overlaps an internal node => traverse.
        bool traverseL = false;
		if (overlapL == 1)
		{
			for(int k= childL->leafStart; k<=childL->leafEnd; k++)
				out[offset + k] = 1;
		}
		else if( overlapL == 2 && !bvh[node->lChild].isLeaf())
		{
			traverseL = true;
		}
        bool traverseR = false;
		if (overlapR == 1)
		{
			for(int k= childR->leafStart; k<=childR->leafEnd; k++)
				out[offset + k] = 1;
		}
		else if( overlapR == 2 && !bvh[node->RChild].isLeaf())
		{
			traverseR = true;
		}

        if (!traverseL && !traverseR)
            node = *--stackPtr; // pop
        else
        {
            node = (traverseL) ? childL : childR;
            if (traverseL && traverseR)
                *stackPtr++ = childR; // push
        }
    }
    while (node != NULL);
}
FORCE_INLINE NIH_DEVICE void FrustumCulling(pyrfrustum_t& frustum, uint32 frustumId,
	Bintree_node* bvh,uint32 priSize,
	uint32* out)
{
	bvhTexHelper helper;
	uint32 offset = priSize*frustumId;
	const uint32 stack_size  = 64;
	uint32 stack[stack_size];
	uint32 top = 0;
	stack[top++] = 0;
	while(top>0)
	{
		uint32 idx = stack[--top];
		//Bintree_node * node = &bvh[idx];
		uint32 RChild = helper.getRChild(idx);
		uint32 LChild = helper.getLChild(idx);
		
		int ret = Intersect(frustum,helper.getBbox(idx));
		if (ret == 2)
		{
			//�ཻ
			
			if(helper.isLeaf(RChild))
			{
				if (Intersect(frustum,helper.getBbox(RChild)))
				{
					out[offset+helper.getPid(RChild)] = 1;					
				}				
			}
			else
				stack[top++] = RChild;

			if (helper.isLeaf(LChild))
			{
				
				if (Intersect(frustum,helper.getBbox(LChild)))
				{
					out[offset+helper.getPid(LChild)] = 1;					
				}				
			}
			else
				stack[top++] = LChild;
		}
		else if (ret == 1)
		{
			//in
			for(int k= helper.getleafStart(idx); k<=helper.getleafEnd(idx);k++)
				out[offset+k] = 1;
		}
	}
}


//FORCE_INLINE NIH_DEVICE void FrustumCulling(pyrfrustum_t& frustum, uint32 frustumId,
//	uint32 priSize,
//	uint32* out)
//{
//	uint32 offset = priSize*frustumId;
//	const uint32 stack_size  = 64;
//	const uint32 nodeOffset = 64;
//	uint32 stack[stack_size];
//	uint32 top = 0;
//	stack[top++] = 0;
//	while(top>0)
//	{
//		uint32 idx = stack[--top];
//		Bintree_node * node = tex1Dfetch(bvhTex,idx*64);
//		int ret = Intersect(frustum,node->box);
//		if (ret == 2)
//		{
//			//�ཻ
//			
//			if(bvh[node->RChild].isLeaf())
//			{
//				if (Intersect(frustum,bvh[node->RChild].box))
//				{
//					out[offset+bvh[node->RChild].pid] = 1;					
//				}				
//			}
//			else
//				stack[top++] = node->RChild;
//
//			if (bvh[node->lChild].isLeaf())
//			{
//				
//				if (Intersect(frustum,bvh[node->lChild].box))
//				{
//					out[offset+bvh[node->lChild].pid] = 1;					
//				}				
//			}
//			else
//				stack[top++] = node->lChild;
//		}
//		else if (ret == 1)
//		{
//			//in
//			for(int k= node->leafStart; k<=node->leafEnd;k++)
//				out[offset+k] = 1;
//		}
//	}
//}
//FORCE_INLINE NIH_DEVICE void FrustumCullingT(pyrfrustum_t& frustum, uint32 frustumId,
//	Bintree_node* bvh,uint32 priSize,
//	uint32* out)
//{
//	Bintree_node* stack[64];
//    Bintree_node** stackPtr = stack;
//    *stackPtr++ = NULL; // push
//	uint32 offset = priSize*frustumId;
//    // Traverse nodes starting from the root.
//    Bintree_node* node = &bvh[0];
//    do
//    {
//        // Check each child node for overlap.
//		Bintree_node* childL = &bvh[node->lChild];
//        Bintree_node* childR = &bvh[node->RChild];
//        int overlapL = ( Intersect(frustum, 
//			childL->box) );
//        int overlapR = (Intersect(frustum, 
//			childR->box) );
//
//        // Query overlaps a leaf node => report collision.
//		if (overlapL>0 && childL->isLeaf())
//			out[offset+ childL->pid] = 1;
//
//		if (overlapR>0 && childR->isLeaf())
//            out[offset+ childR->pid] = 1;
//
//		/*if (overlapL == 1 && !childL->isLeaf)
//		{
//			for(int k=childL->leafStart; k<=childL->leafEnd;k++)
//				out[offset+k] = 1;
//		}
//
//		if (overlapR == 1 && !childR->isLeaf )
//		{
//			for(int k=childR->leafStart; k<=childR->leafEnd;k++)
//				out[offset+k] = 1;
//		}*/
//        // Query overlaps an internal node => traverse.
//        bool traverseL = (overlapL == 2 && !childL->isLeaf());
//        bool traverseR = (overlapR ==2 && !childR->isLeaf());
//
//        if (!traverseL && !traverseR)
//            node = *--stackPtr; // pop
//        else
//        {
//            node = (traverseL) ? childL : childR;
//            if (traverseL && traverseR)
//                *stackPtr++ = childR; // push
//        }
//    }
//    while (node != NULL);
//}
__global__ void BruteforceFrustumCullingKernel(pyrfrustum_t* frustum, Bbox3f* boxes, uint32 priSize,uint32* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < priSize; 
		i += step) 
	{
		int offset = i*6;
		float p[6];
		p[0] = tex1Dfetch(BboxTex,offset);
		p[1] = tex1Dfetch(BboxTex,offset+1);
		p[2] = tex1Dfetch(BboxTex,offset+2);
		p[3] = tex1Dfetch(BboxTex,offset+3);
		p[4] = tex1Dfetch(BboxTex,offset+4);
		p[5] = tex1Dfetch(BboxTex,offset+5);
		
	
		if (Intersect(*frustum,p) >0 )
			list[i] = 1;
	}
}
__global__ void FrustumCullingKernel(pyrfrustum_t* frustumP, int frustum_num, Bintree_Node* bvh, uint32 priSize,uint32* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < frustum_num; 
		i += step) 
	{
		pyrfrustum_t frustum = frustumP[i];
		FrustumCulling(frustum,i,bvh,priSize,list);
		//FrustumCullingT(frustum,i,bvh,priSize,list);
	}
}
__global__ void FrustumCullingKernel(pyrfrustum_t* frustumP, int frustum_num, Bintree_node* bvh, uint32 priSize,uint32* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < frustum_num; 
		i += step) 
	{
		pyrfrustum_t frustum = frustumP[i];
		FrustumCulling(frustum,i,bvh,priSize,list);
		//FrustumCullingT(frustum,i,bvh,priSize,list);
	}
}
__global__ void FrustumCullingKernel(pyrfrustum_t* frustumP, int frustum_num, Bintree* bvh, uint32 priSize,uint32* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < frustum_num; 
		i += step) 
	{
		pyrfrustum_t frustum = frustumP[i];
		FrustumCulling(frustum,i,bvh,priSize,list);
		//FrustumCullingT(frustum,i,bvh,priSize,list);
	}
}

__global__ void FrustumCullingKernel(pyrfrustum_t* frustumP, int frustum_num, cuda::DBVH* bvh, uint32 priSize,uint32* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < frustum_num; 
		i += step) 
	{
		pyrfrustum_t frustum = frustumP[i];
		FrustumCulling(frustum,i,bvh,priSize,list);
	}
}
//__global__ void CollisonDetectionKernel(cuda::DBVH* bvh, cuda::SimpleList<Pair>* list, int size)
//{
//	int idx = threadIdx.x + blockDim.x * blockIdx.x;
//    if (idx < size)
//    {
//        Bbox3f qbox = bvh->leafBoxes[idx];
//		uint32 qId = bvh->leaves[idx].pid;
//        traverseIterative(qbox,qId, idx, bvh,list); 
//                      
//    }
//}



void CpuKlbvhTest(const thrust::host_vector<Vector3f>& h_points,
	const thrust::host_vector<Bbox3f>& h_boxes, const Bbox3f& gbox,
	pyrfrustum_t& frustum, BVH& bvh);
void GpuKlbvhTest(const thrust::host_vector<Vector3f>& h_points,
	const thrust::host_vector<Bbox3f>& h_boxes, 
	const Bbox3f& gBox,
	pyrfrustum_t& frustum, 
	thrust::host_vector<Bvh_Node>& h_nodes,
	thrust::host_vector<Bvh_Node>& h_leaves,
	thrust::host_vector<Bbox3f>& h_nodeBoxes,
	thrust::host_vector<Bbox3f>& h_leafBoxes
	)
{
	
	size_t size = h_points.size();
	thrust::device_vector<Vector3f> d_points = h_points;
	thrust::device_vector<Bbox3f> d_boxes = h_boxes;

	//{
	//random data
	/*size = 12;
	thrust::host_vector<Vector3f> points(size);
	thrust::host_vector<Bbox3f> boxes(size);

	for(int i =0; i< points.size(); i++)
	{
	points[i] = Vector3f(1.0*(rand()%100)/100,1.0*(rand()%100)/100,1.0*(rand()%100)/100);
	printVector3f(points[i]);
	boxes[i].insert(points[i]);
	gBox.insert(boxes[i]);
	}	
	d_points = points;
	d_boxes = boxes;*/
	//}
	//load from obj file
	/*size = loadObj(objFileName,d_points,d_boxes,gBox);*/
	thrust::device_vector<Bvh_Node> nodes(size-1);
	thrust::device_vector<Bvh_Node> leaves(size);
	hipcub::CachingDeviceAllocator allocator(true);
	cuda::KBvh_Builder builder(nodes,leaves,allocator);

	/*hipcub::DoubleBuffer<uint32> d_codes;
	size_t n_points = points.size();
	allocator.DeviceAllocate((void**)&d_codes.d_buffers[0], sizeof(uint32) * n_points);
	thrust::device_ptr<uint32> d_ptr(d_codes.d_buffers[0]);
	thrust::transform(
	d_points.begin(),
	d_points.begin() + n_points,
	d_ptr,
	morton_functor<uint32>( gBox ) );
	uint32 * h_codes = (uint32*)malloc(sizeof(uint32)*n_points);
	hipMemcpy(h_codes,d_codes.d_buffers[0],sizeof(uint32)*n_points,hipMemcpyDeviceToHost);
	for(int i=0; i<n_points; i++)
	std::cout<<h_codes[i]<<std::endl;*/
	cuda::DBVH h_bvh;
	/*hipMalloc((void**)&d_bvh,sizeof(cuda::DBVH));*/
	GpuTimer timer;
	timer.Start();
	builder.build(gBox,d_points.begin(),d_points.end(),d_boxes.begin(),d_boxes.end(),&h_bvh);
	timer.Stop();
	std::cout<<"build time "<<timer.ElapsedMillis()<<" ms"<<std::endl;

	hipBindTexture( NULL, BboxTex,
		builder.getLeafBoxes(),
		sizeof(Bbox3f)*size );

	h_nodes = nodes;
	h_leaves = leaves;	


	//for(int i = 0; i<h_nodes.size(); i++)
	//{ 
	//	std::cout<<" parent idx is "<<h_nodes[i].parentIdx<<" ,";
	//	
	//	if(h_nodes[i].l_isleaf)
	//	{
	//		std::cout<<i<<" left child "<<" is leaf "<<h_nodes[i].getChild(0);
	//	}
	//	else
	//	{
	//		std::cout<<i<<" left child "<<" is internal "<<h_nodes[i].getChild(0);				

	//	}
	//	if(h_nodes[i].r_isleaf)
	//	{
	//		std::cout<<" right child "<<" is leaf "<<h_nodes[i].getChild(1)<<std::endl;
	//	}
	//	else
	//	{
	//		std::cout<<" right child "<<" is internal "<<h_nodes[i].getChild(1)<<std::endl;
	//	}
	//}
	//for(int i=0; i<h_leaves.size(); i++)
	//{
	//	std::cout<<i<<" parent is "<<h_leaves[i].parentIdx<<std::endl;
	//	std::cout<<" pid is "<<h_leaves[i].pid<<std::endl;
	//}
	
	h_nodeBoxes.resize(size-1);
	Bbox3f* p_nodeBoxes = thrust::raw_pointer_cast(&h_nodeBoxes.front());
	hipMemcpy(p_nodeBoxes,builder.getNodeBoxes(),sizeof(Bbox3f)*(size-1),hipMemcpyDeviceToHost);

	h_leafBoxes.resize(size);
	Bbox3f* p_leafBoxes = thrust::raw_pointer_cast(&h_leafBoxes.front());
	hipMemcpy(p_leafBoxes,builder.getLeafBoxes(),sizeof(Bbox3f)*(size),hipMemcpyDeviceToHost);




	int frustumCount = 1;
	thrust::host_vector<pyrfrustum_t> h_frustums(frustumCount);
	for(int i=0; i< frustumCount; i++)
		h_frustums[i] = frustum;

	const uint32 BLOCK_SIZE = 128;
	int numSMs;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
	size_t max_blocks = 65535;
	size_t n_blocks   = nih::min( max_blocks, (frustumCount + (BLOCK_SIZE*numSMs)-1) / (BLOCK_SIZE*numSMs) );
	thrust::device_vector<pyrfrustum_t> d_frustums = h_frustums;

	Bvh_Node* d_nodes = thrust::raw_pointer_cast(&(nodes.front()));
	Bvh_Node* d_leaves = thrust::raw_pointer_cast(&(leaves.front()));
	cuda::DBVH* d_bvh = NULL;
	hipMalloc((void**)&d_bvh,sizeof(BVH));
	hipMemcpy(d_bvh,&h_bvh,sizeof(BVH),hipMemcpyHostToDevice);
	uint32* d_list;
	thrust::device_vector<uint32> d_vector(size*frustumCount,0);
	d_list = thrust::raw_pointer_cast(&d_vector.front());
	timer.Start();
	FrustumCullingKernel<<<n_blocks*numSMs,BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_frustums.front()),frustumCount, d_bvh, size,d_list);
	timer.Stop();
	std::cout<<"culling time "<<timer.ElapsedMillis()<<" ms"<<std::endl;
	thrust::host_vector<uint32> h_list = d_vector;
	uint32 k = 0;
	for(int i=0; i<size*frustumCount; i++)
		if(h_list[i] == 1)
			k++;
	std::cout<<"total "<<h_list.size()<<" in "<<k<<std::endl;
	
	
	
	Bintree_node* nbvh;
	uint32 nbvh_size = h_nodes.size()+h_leaves.size();
	nbvh = new Bintree_node[nbvh_size];
    Timer ctimer;
	ctimer.start();
	cuda::DFSBintree(&h_nodes,&h_leaves,&h_nodeBoxes,&h_leafBoxes,nbvh);
	//cuda::BFSBintree(&h_nodes,&h_leaves,&h_nodeBoxes,&h_leafBoxes,nbvh);
	ctimer.stop();
	std::cout<<"build DFSBintree time "<<ctimer.seconds()*1000<<"ms"<<std::endl;
	Bintree_node* d_nbvh;
	hipMalloc((void**)&d_nbvh,sizeof(Bintree_node)*nbvh_size);
	hipMemcpy(d_nbvh,nbvh,sizeof(Bintree_node)*nbvh_size,hipMemcpyHostToDevice);
	hipBindTexture( NULL, bvhTex,
		d_nbvh, sizeof(Bintree_node)*nbvh_size );

	thrust::device_vector<uint32> d_vector1(size*frustumCount,0);
	d_list = thrust::raw_pointer_cast(&d_vector1.front());
	timer.Start();
	FrustumCullingKernel<<<n_blocks*numSMs,BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_frustums.front()),frustumCount, d_nbvh, size,d_list);
	timer.Stop();
	std::cout<<"dfs bvh culling time "<<timer.ElapsedMillis()<<" ms"<<std::endl;
	h_list = d_vector1;
	k = 0;
	for(int i=0; i<size*frustumCount; i++)
		if(h_list[i] == 1)
			k++;
	std::cout<<"total "<<h_list.size()<<" in "<<k<<std::endl;
	hipFree(d_nbvh);
	delete[] nbvh;

	Bintree_node * Nbvh;
	Nbvh = new Bintree_node[nbvh_size];   
	ctimer.start();
	cuda::BFSBintree(&h_nodes,&h_leaves,&h_nodeBoxes,&h_leafBoxes,Nbvh);
	ctimer.stop();
	std::cout<<"build BFSBintree time "<<ctimer.seconds()*1000<<"ms"<<std::endl;
	Bintree_node* d_Nbvh;
	hipMalloc((void**)&d_Nbvh,sizeof(Bintree_node)*nbvh_size);
	hipMemcpy(d_Nbvh,Nbvh,sizeof(Bintree_node)*nbvh_size,hipMemcpyHostToDevice);
	thrust::device_vector<uint32> d_vector0(size*frustumCount,0);
	d_list = thrust::raw_pointer_cast(&d_vector0.front());
	timer.Start();
	FrustumCullingKernel<<<n_blocks*numSMs,BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_frustums.front()),frustumCount, d_Nbvh, size,d_list);
	timer.Stop();
	std::cout<<"BFS Nbvh culling time "<<timer.ElapsedMillis()<<" ms"<<std::endl;
	h_list = d_vector0;
	k = 0;
	for(int i=0; i<size*frustumCount; i++)
		if(h_list[i] == 1)
			k++;
	std::cout<<"total "<<h_list.size()<<" in "<<k<<std::endl;
	hipFree(d_Nbvh);
	delete[] Nbvh;



	//Bintree h_bvhSoa;
	//cuda::DFSBintreeSOA(&h_nodes,&h_leaves,&h_nodeBoxes,&h_leafBoxes,&h_bvhSoa);
	//Bintree* d_bvhSoa;
	//hipMalloc((void**)&d_bvhSoa,sizeof(Bintree));
	//hipMemcpy(d_bvhSoa,&h_bvhSoa,sizeof(Bintree),hipMemcpyHostToDevice);
	//thrust::device_vector<uint32> d_vector3(size*frustumCount,0);
	//d_list = thrust::raw_pointer_cast(&d_vector3.front());
	//timer.Start();
	//FrustumCullingKernel<<<n_blocks*numSMs,BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_frustums.front()),frustumCount, d_bvhSoa, size,d_list);
	//timer.Stop();
	//std::cout<<"dfs bvh soa culling time "<<timer.ElapsedMillis()<<" ms"<<std::endl;
	//h_list = d_vector3;
	//k = 0;
	//for(int i=0; i<size*frustumCount; i++)
	//	if(h_list[i] == 1)
	//		k++;
	//std::cout<<"total "<<h_list.size()<<" in "<<k<<std::endl;
	//hipFree(h_bvhSoa.boxPtr);
	//hipFree(h_bvhSoa.isLeafPtr);
	//hipFree(h_bvhSoa.LChildPtr);
	//hipFree(h_bvhSoa.leafEndPtr);
	//hipFree(h_bvhSoa.leafStartPtr);
	//hipFree(h_bvhSoa.pidPtr);
	//hipFree(h_bvhSoa.RChildPtr);
	//hipFree(d_bvhSoa);

	thrust::device_vector<uint32> d_vector2(size,0);
	d_list = thrust::raw_pointer_cast(&d_vector2.front());
	n_blocks   = nih::min( max_blocks, (size + (BLOCK_SIZE*numSMs)-1) / (BLOCK_SIZE*numSMs) );
	timer.Start();
	BruteforceFrustumCullingKernel<<<n_blocks*numSMs,BLOCK_SIZE>>>(thrust::raw_pointer_cast(&d_frustums.front()), builder.getLeafBoxes(), size,d_list);
	timer.Stop();
	std::cout<<"brute force culling time "<<timer.ElapsedMillis()<<" ms"<<std::endl;
	h_list = d_vector2;
	k = 0;
	for(int i=0; i<size; i++)
		if(h_list[i] == 1)
			k++;
	std::cout<<"total "<<h_list.size()<<" in "<<k<<std::endl;




	/*cuda::SimpleList<Pair> h_plist(h_points.size()*2);
	cuda::SimpleList<Pair>* d_plist=NULL;
	hipMalloc((void**)(&d_plist),sizeof(cuda::SimpleList<Pair>));
	hipMemcpy(d_plist,&h_plist,sizeof(cuda::SimpleList<Pair>),hipMemcpyHostToDevice);
	const size_t blocks   = nih::min( max_blocks, (size + (BLOCK_SIZE*numSMs)-1) / (BLOCK_SIZE*numSMs) );
	timer.Start();
	CollisonDetectionKernel<<<blocks*numSMs,BLOCK_SIZE>>>(d_bvh,d_plist,size);
	timer.Stop();
	std::cout<<"collison detec time "<<timer.ElapsedMillis()<<" ms"<<std::endl;
	hipMemcpy(&h_plist,d_plist,sizeof(cuda::SimpleList<uint32>),hipMemcpyDeviceToHost);
	std::cout<<"overlapped "<<h_plist.size()<<std::endl;
*/
	//hipFree(d_bvh);
}

bool BboxCompare(const Bbox3f& lbox, const Bbox3f& rbox)
{
	const double zero = 0.0001;

	return (abs(lbox.m_min[0]-rbox.m_min[0])<zero &&
		abs(lbox.m_min[1]-rbox.m_min[1])<zero && 
		abs(lbox.m_min[2]-rbox.m_min[2])<zero &&
		abs(lbox.m_max[0]-rbox.m_max[0])<zero &&
		abs(lbox.m_max[1]-rbox.m_max[1])<zero && 
		abs(lbox.m_max[2]-rbox.m_max[2])<zero );
}

int main(int argc, char** argv)
{
	std::cout<<sizeof(Bintree_node)<<std::endl;
	char* fileName ="testbox.obj";
	if (argc == 2)
	{
		fileName = (argv[1]);
	}
	// Projection matrix : 45�� Field of View, 4:3 ratio, display range : 0.1 unit <-> 100 units
	glm::mat4 Projection = glm::perspective(45.0f, 4.0f / 3.0f, 0.01f, 50.0f);
	// Camera matrix
	glm::mat4 View       = glm::lookAt(
		glm::vec3(0,0,-7), // Camera is at (4,3,3), in World Space
		glm::vec3(0,0,0), // and looks at the origin
		glm::vec3(0,1,0)  // Head is up (set to 0,-1,0 to look upside-down)
		);
	// Model matrix : an identity matrix (model will be at the origin)
	glm::mat4 Model      = glm::mat4(1.0f);  // Changes for each model !

	// Our ModelViewProjection : multiplication of our 3 matrices
	glm::mat4 MVP        = Projection * View * Model; // Remember, matrix multiplication is the other way around

	Matrix4x4 mvp;		
	memcpy(&mvp,&MVP[0][0],16*sizeof(float));
	pyrfrustum_t frustum;
	ExtractPlanesGL(frustum.planes,mvp,true);

	thrust::host_vector<Vector3f> h_points;
	thrust::host_vector<Bbox3f> h_boxes;
	Bbox3f gBox;
	loadObj(fileName,h_points,h_boxes,gBox);
	//loadRandom(3650,h_points,h_boxes,gBox);

	thrust::host_vector<Bvh_Node> h_nodes,h_leaves;
	thrust::host_vector<Bbox3f> h_nodeBoxes,h_leafBoxes;
	std::cout<<"gpu:"<<std::endl;
	GpuKlbvhTest(h_points,h_boxes,gBox,frustum,h_nodes,h_leaves,h_nodeBoxes,h_leafBoxes);
	std::cout<<"cpu:"<<std::endl;
	BVH cpuBvh;
	CpuKlbvhTest(h_points,h_boxes,gBox,frustum,cpuBvh);

	std::cout<<"�Ƚ�"<<std::endl;
	if (cpuBvh.nodes.size() != h_nodes.size())
	{
		std::cout<<"size is different!"<<std::endl;
	}
	for (int i=0; i<cpuBvh.nodes.size(); i++)
	{
		if (!cpuBvh.nodes[i].equal(h_nodes[i]))
		{
			std::cout<<"node "<<i<<"is different"<<std::endl;
			break;
		}
		if (cpuBvh.leafs[i].parentIdx != h_leaves[i].parentIdx)
		{
			std::cout<<"leaf "<<i<<"is different"<<std::endl;
			std::cout<<"cpu "<<cpuBvh.leafs[i].parentIdx<<" gpu "<<h_leaves[i].parentIdx<<std::endl;
			break;
		}
		if (!BboxCompare(cpuBvh.node_Boxes[i],h_nodeBoxes[i]))
		{
			std::cout<<"node box "<<i<<"is different"<<std::endl;
			break;
		}
		if (!BboxCompare(cpuBvh.leaf_Boxes[i],h_leafBoxes[i]))
		{
			std::cout<<"leaf box "<<i<<"is different"<<std::endl;
			std::cout<<"cpu ";
			printBbox3f(cpuBvh.leaf_Boxes[i]);
			std::cout<<"gpu ";
			printBbox3f(h_leafBoxes[i]);
			break;
		}
	}

	return 0;
}
